#include "hip/hip_runtime.h"
/****** BLUR effect *******/
#include <stdio.h>
#include <math.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
//For image loading and writing
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image/stb_image_write.h"

/*****************************************************************************
/*kernel and device functions
*****************************************************************************/

__device__ int mirror(int a, int N){
    if(a < 0) return -a - 1;
    else if(a > N - 1) return N + N - 1 - a;
    else return a;
}


__global__ void blurKernel(unsigned char *blur, unsigned char *img, int width,int height,int channels,int kS, int totalThreads)
{       
 
  int chunkSize = (int)floorf(width/totalThreads);
  int index = (blockDim.x * blockIdx.x) + threadIdx.x;
  int start = chunkSize * index;
  int end = start + (chunkSize - 1);
  if(index == totalThreads-1) end = width;

  int k = (kS - 1) / 2;
 
  for (int cx = start; cx <= end; cx ++){
        for (int cy = 0; cy < height; cy ++){
            int R = 0;
            int G = 0;
            int B = 0;

            for( int j = -k; j<= k ; j++){
                
                int sR = 0;
                int sG = 0;
                int sB = 0;

                int ny = mirror(cy + j, height);
                
                for( int i = -k; i<= k ; i++){                    
                    int nx = mirror(cx + i, width);
                    //MODIFICAR, LECTURA DE LA IMAGEN
                    sR+= (uint8_t) *(img + channels*( nx + ny*width ));
                    sG+= (uint8_t) *(img + channels*( nx + ny*width ) + 1);
                    sB+= (uint8_t) *(img + channels*( nx + ny*width ) + 2);                
                }
                R+= sR/kS;
                G+= sG/kS;
                B+= sB/kS;
            }
            //MODIFICAR, ESCRITURA DE LA IMAGEN
            *(blur + channels*( cx + cy*width ))       = R/kS;
            *(blur + channels*( cx + cy*width ) + 1)   = G/kS;
            *(blur + channels*( cx + cy*width ) + 2)   = B/kS;
        }
    }
  
}


/******************************************************************************/
/*Host code
*******************************************************************************/

int main(int argc,char *argv[])
{   
    /*device info*/
    int maxNumTh;
    int numBlocks;
    int device;   
    hipGetDevice ( &device );
    hipDeviceGetAttribute ( &maxNumTh, hipDeviceAttributeMaxThreadsPerBlock, device);
    hipDeviceGetAttribute ( &numBlocks, hipDeviceAttributeMultiprocessorCount, device);
    printf("numTh: %i numBlocks: %i \n",maxNumTh,numBlocks);


    //****Lectura de argumentos***//
    char *inputFile = argv[1];
    char *outputFile = argv[2];
    int kS = atoi(argv[3]); //Kernel size
    //int threadNum; GLOBAL
    //threadNum = atoi(argv[4]);

    //****Lectura imagen y reserva de memoria en el host*****/
    unsigned char *img, *blur;
    int w,h,channels;
    img = stbi_load(inputFile, &w, &h, &channels, 0);
    if(img == NULL){
        printf("Error leyendo imagen");
        exit(1);
    }

    size_t img_size = w * h * channels;   //TAMAÑO IMAGEN 
    blur = (unsigned char *)malloc(img_size);
    if(blur == NULL){
        printf("Error creando nueva imagen (malloc blur) (F)");
        exit(1);
    }

    /****Reserva de espacio en dispositivo****/
    unsigned char *d_img, *d_blur;

    hipError_t err = hipSuccess;
    err = hipMalloc((void **)&d_img,img_size);
    if(err != hipSuccess){
        fprintf(stderr, "Error reservando memoria para img en dispositivo (codigo de error %s)!'n'",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_blur,img_size);
    if(err != hipSuccess){
        fprintf(stderr, "Error reservando memoria para img en dispositivo (codigo de error %s)!'n'",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /***Copiado de Img en dispositivo*****/
    err = hipMemcpy(d_img,img,img_size,hipMemcpyHostToDevice);
    if(err != hipSuccess){
        fprintf(stderr, "Error copiando img a dispositivo (codigo de error %s)!'n'",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /****Lanzamiento de kernel*****/
    int NUMTHREADS = numBlocks*maxNumTh;
    int threadsPerBlock = NUMTHREADS/numBlocks;
    int totalThreads = numBlocks * threadsPerBlock;
    printf("CUDA kernel lanzado con %d blocks of %d threads Total: %i       ", numBlocks, threadsPerBlock, totalThreads  );
    blurKernel<<<numBlocks,threadsPerBlock>>>(d_blur,d_img,w,h,channels,kS,totalThreads);
    err = hipGetLastError();
    if(err != hipSuccess){
        fprintf(stderr, "Error lanzando blur kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /***Copia de Blur de dispositivo a host****/
    err = hipMemcpy(blur,d_blur,img_size,hipMemcpyDeviceToHost);
    if(err != hipSuccess){
        fprintf(stderr, "Error copiando blur de dispositivo a host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /***Liberado de memoria de dispositivo***/
    err = hipFree(d_img);
    if (err != hipSuccess){
        fprintf(stderr, "Error liberando espacio de d_img (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_blur);
    if (err != hipSuccess){
        fprintf(stderr, "Error liberando espacio de d_blur (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /****Guardado de la imagen en archivo y liberado de memoria****/
    stbi_write_jpg(outputFile, w, h, channels, blur, 100);
    stbi_image_free(img);
    free(blur);

    err = hipDeviceReset();
    if (err != hipSuccess){
        fprintf(stderr, "Error reiniciando dispositivo! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
        
    return 0;
}